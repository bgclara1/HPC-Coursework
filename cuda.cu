/**
 * @file cuda.cu
 * @brief  A solver for particle interaction in an enclosed space modelled using Lennard-Jones potential using parallelisation with CUDA.
 *
 * This program simulates the interaction of particles within an enclosed space. 
 * It uses Lennard-Jones potential to model the interactions. 
 * This programme uses a parallel OMP algorithm to solve the problem. 
 * The programme allows the user to specify the dimensions of the box, the number of particles, the ratio of heavy to light particles,
 * the temperature, the duration of the simulation and the timestep. 
 * The user may also chose to run 6 example simulations.
 */


 #include <iostream>
 #include <map>
 #include <string>
 #include <vector>
 #include <cmath>
 #include <cstdlib>
 #include <fstream>
 #include <iomanip>
 #include <ctime>
 #include <chrono>
 #include <hip/hip_runtime.h>
 
 using namespace std;
 
 __constant__ double s6_table[4] = {0.0, 1.0, 64.0, 729.0}; //The values of sigma ^ 6 precalculated and set as a global constant as 
 //the variable never changes and is computationally intensive.
 
 
 /**
  * @brief Allocates and initializes unified memory for simulation arrays.
  *
  * This function allocates managed memory for the updating variables X,Y etc...,
  * Then the memory block contents is set to zero.
  *
  * @param totalSteps Number of simulation steps         
  * @param numParticles Number of particles
  * @param X Pointer to the pointer for x coordinates
  * @param Y Pointer to the pointer for y coordinate
  * @param Z Pointer to the pointer for z coordinates
  * @param U Pointer to the pointer for x velocities
  * @param V Pointer to the pointer for y velocities
  * @param W Pointer to the pointer for z velocities
  * @param E Pointer to the pointer that will hold the kinetic energy
  * @param speed Pointer to the pointer that will hold the speeds
  * @param Fx Pointer to the pointer of the x direction force component of each particle
  * @param Fy Pointer to the pointer of the y direction force component of each particle
  * @param Fz Pointer to the pointer of the z direction force component of each particle
  * @param type Pointer to the pointer that will hold the particle types.
  */
 void variableInitialisation(int totalSteps, int numParticles,
     double** X, double** Y, double** Z,
     double** U, double** V, double** W,
     double** E, double** speed,
     double** Fx, double** Fy, double** Fz,
     double** type)
 {
     double size = totalSteps * numParticles * sizeof(double);
     hipMallocManaged((void**)X, size);  // basically allocate memory and put address in the pointer that X points to
     hipMallocManaged((void**)Y, size);
     hipMallocManaged((void**)Z, size);
     hipMallocManaged((void**)U, size);
     hipMallocManaged((void**)V, size);
     hipMallocManaged((void**)W, size);
     hipMallocManaged((void**)E, size);
     hipMallocManaged((void**)speed, size);
     hipMallocManaged((void**)Fx, size);
     hipMallocManaged((void**)Fy, size);
     hipMallocManaged((void**)Fz, size);
 
     hipMemset(*X, 0, size);
     hipMemset(*Y, 0, size);
     hipMemset(*Z, 0, size);
     hipMemset(*U, 0, size);
     hipMemset(*V, 0, size);
     hipMemset(*W, 0, size);
     hipMemset(*E, 0, size);
     hipMemset(*speed, 0, size);
     hipMemset(*Fx, 0, size);
     hipMemset(*Fy, 0, size);
     hipMemset(*Fz, 0, size);
 
     double typeSize = numParticles * sizeof(double);
     hipMallocManaged((void**)type, typeSize);
     hipMemset(*type, 0, typeSize);
 }
 
 
 /**
  * @brief Initializes particles with random positions and velocities for when the user inputs --ic-random as a command line parameter.
  *
  * This function assigns random initial positions and velocities to particles. The posititions must be within the default (20) or requested dimensions
  * of the container. The velocities are generated between -0.5 and 0.5.
  * Additionally, it randomly assigns particle types according to the default (10%) percentage of heavy particles or the percentage requested by the user.
  *
  * @param numParticles Number of particles
  * @param Lx Length of the container in the x direction
  * @param Ly Length of the container in the y direction
  * @param Lz Length of the container in the z direction
  * @param percent_type1 Percentage of particles of type 1 (heavy)
  * @param X Pointer to x coordinates of each particle
  * @param Y Pointer to y coordinates of each particle
  * @param Z Pointer to z coordinates of each particle
  * @param U Pointer to x velocities of each particle
  * @param V Pointer to y velocities of each particle
  * @param W Pointer to z velocities of each particle
  * @param type Pointer to an array of the type of each particle (0 or 1 ie. light or heavy)
  */
 void icRandom(int numParticles, double Lx, double Ly, double Lz, double percent_type1,
     double* X, double* Y, double* Z,
     double* U, double* V, double* W,
     double* type)
 {
     srand(time(0));
 
     // Initialize positions and velocities.
     for (int i = 0; i < numParticles; i++) {
         double cx, cy, cz;
         while (true) {
             cx = ((double)rand() / RAND_MAX) * Lx;
             cy = ((double)rand() / RAND_MAX) * Ly;
             cz = ((double)rand() / RAND_MAX) * Lz;
             bool valid = true;
             for (int j = 0; j < i; j++) {
                 double dx = cx - X[j];
                 double dy = cy - Y[j];
                 double dz = cz - Z[j];
                 if (dx * dx + dy * dy + dz * dz < 0.25) {  // 0.5^2 = 0.25
                     valid = false;
                     break;
                 }
             }
             if (valid)
                 break;
         }
         X[i] = cx;
         Y[i] = cy;
         Z[i] = cz;
         U[i] = ((double)rand() / RAND_MAX) - 0.5;
         V[i] = ((double)rand() / RAND_MAX) - 0.5;
         W[i] = ((double)rand() / RAND_MAX) - 0.5;
     }
 
     int numType1 = (int)ceil(numParticles * (percent_type1 / 100.0));
     for (int i = 0; i < numParticles; i++) {
         type[i] = 0;
     }
     for (int i = 0; i < numType1; i++) {
         type[i] = 1;
     }
     for (int i = 0; i < numParticles; i++) {
         int j = rand() % numParticles;
         double temp = type[i];
         type[i] = type[j];
         type[j] = temp;
     }
 }
 
 
 /**
  * @brief Fetches predefined test cases.
  *
  * The brief specifies six ecample cases. This function generates a map of the paramaters for each of the examples.
  *
  * @return A map where keys are test case names (eg. --ic-one-vel) and their values are maps containing their respective simulation parameters.
  */
 map<string, map<string, vector<double>>> getTestCases() {
     map<string, map<string, vector<double>>> testCaseDict;
     testCaseDict["--ic-one"] = {
         {"runtime", {0.2}},
         {"numParticles", {1}},
         {"x", {10.0}},
         {"y", {10.0}},
         {"z", {10.0}},
         {"u", {0.0}},
         {"v", {0.0}},
         {"w", {0.0}},
         {"type", {0}}
     };
     testCaseDict["--ic-one-vel"] = {
         {"runtime", {20.0}},
         {"numParticles", {1}},
         {"x", {10.0}},
         {"y", {10.0}},
         {"z", {10.0}},
         {"u", {5.0}},
         {"v", {2.0}},
         {"w", {1.0}},
         {"type", {0}}
     };
     testCaseDict["--ic-two"] = {
         {"runtime", {50}},
         {"numParticles", {2}},
         {"x", {8.5, 11.5}},
         {"y", {10.0, 10.0}},
         {"z", {10.0, 10.0}},
         {"u", {0.0, 0.0}},
         {"v", {0.0, 0.0}},
         {"w", {0.0, 0.0}},
         {"type", {0, 0}}
     };
     testCaseDict["--ic-two-pass1"] = {
         {"runtime", {50.0}},
         {"numParticles", {2}},
         {"x", {8.5, 11.5}},
         {"y", {11.5, 8.5}},
         {"z", {10.0, 10.0}},
         {"u", {0.5, -0.5}},
         {"v", {0.0, 0.0}},
         {"w", {0.0, 0.0}},
         {"type", {0, 0}}
     };
     testCaseDict["--ic-two-pass2"] = {
         {"runtime", {50.0}},
         {"numParticles", {2}},
         {"x", {8.5, 11.5}},
         {"y", {11.3, 8.7}},
         {"z", {10.0, 10.0}},
         {"u", {0.5, -0.5}},
         {"v", {0.0, 0.0}},
         {"w", {0.0, 0.0}},
         {"type", {0, 0}}
     };
     testCaseDict["--ic-two-pass3"] = {
         {"runtime", {50.0}},
         {"numParticles", {2}},
         {"x", {8.5, 11.5}},
         {"y", {11.3, 8.7}},
         {"z", {10.0, 10.0}},
         {"u", {0.5, -0.5}},
         {"v", {0.0, 0.0}},
         {"w", {0.0, 0.0}},
         {"type", {1, 1}}
     };
     return testCaseDict;
 }
 
 
 
 /**
  * @brief CUDA kernel to update variables for particles.
  *
  * This kernel updates the positions, velocities, energies, and forces on particles based on
  * Lennard-Jones potential equations. Boundary conditions are applied
  * to keep particles within the simulation box. If a temperature is set by the user it is enforced at this point.
  * 
  * It uses CUDA GPU parallelisation to improve the runtime of complex particle simulations.
  *
  * @param min_dist Minimum distance between any two particles
  * @param dt Time step
  * @param numParticles Number of particles
  * @param Lx Length of the container in the x direction
  * @param Ly Length of the container in the y direction
  * @param Lz Length of the container in the z direction
  * @param type Pointer to an array of the type of each particle (0 or 1 ie. light or heavy)
  * @param temperature Chosen simulation temperature
  * @param tempProvided Boolean indicating if the temperature is provided
  * @param kb Boltzmann constant
  * @param epsilon Lennard-Jones Potential epsilon values
  * @param sigma Lennard-Jones Potential sigma values
  * @param X Pointer to x coordinates of each particle
  * @param Y Pointer to y coordinates of each particle
  * @param Z Pointer to z coordinates of each particle
  * @param U Pointer to x velocities of each particle
  * @param V Pointer to y velocities of each particle
  * @param W Pointer to z velocities of each particle
  * @param E Pointer to the pointer that will hold the kinetic energies of each particle
  * @param speed Pointer to the speeds
  * @param Fx Pointer to the x direction force component of each particle
  * @param Fy Pointer to the y direction force component of each particle
  * @param Fz Pointer to the z direction force component of each particle
  */
 __global__   //CUDA kernal updateVars
 void updateVars(int numParticles, double dt, double Lx, double Ly, double Lz,
     double* type, double temperature, bool tempProvided, double kb,
     const int epsilon[2][2], const int sigma[2][2],
     double* X, double* Y, double* Z,
     double* U, double* V, double* W,
     double* E, double* speed, double* Fx, double* Fy, double* Fz)
 {
 
     
     //global thread index = thread id + num threads per block + current block index
     int tid = threadIdx.x + blockDim.x * blockIdx.x; 
     if (tid >= numParticles) return; // exit condition if thread index pointing to a particle that doesn't exist
     for (int i = 0; i < numParticles; i++) {
         for (int j = i + 1; j < numParticles; j++) {
             double xij = X[i] - X[j];
             double yij = Y[i] - Y[j];
             double zij = Z[i] - Z[j];
             double rij = xij*xij + yij*yij + zij*zij; // r squared
             int t1 = type[i];
             int t2 = type[j];
             int e = epsilon[t1][t2];  // finds e and s of the particular particle pair
             int s = sigma[t1][t2];
 
             double inv_r4 = 1.0 / (rij * rij * rij * rij); //calculation split up and not including pow for optimisation reasons
             double sigma6 = s6_table[s] * inv_r4;
             double sigma12 = sigma6 * sigma6 * rij;
             double coeff = -24.0 * e * (2.0 * sigma12 - sigma6);
 
             Fx[i] -= xij * coeff;       //calculate the net forces
             Fy[i] -= yij * coeff;
             Fz[i] -= zij * coeff;
             Fx[j] += xij * coeff;   // the opposite sign is applied to the j indices as this represents the missing lower triangle of the matrix
             Fy[j] += yij * coeff;
             Fz[j] += zij * coeff;
         }
     }
     
     for (int i = 0; i < numParticles; i++) {
         int m = (type[i] == 0) ? 1 : 10; // if true pick 1 else 10
         U[i] += dt * Fx[i] / m; //update velocities
         V[i] += dt * Fy[i] / m;
         W[i] += dt * Fz[i] / m;
     }
 
     double E_total = 0.0;
     for (int i = 0; i < numParticles; i++) {        // calculate kinetic energy
         int m = (type[i] == 0) ? 1 : 10;
         double speed2 = U[i]*U[i] + V[i]*V[i] + W[i]*W[i];
         E[i] = 0.5 * m * speed2;
         E_total += E[i];
     }
 
     if (tempProvided) {                                                      //update velocity if temperature is defined by the user
         double currentTemp = (2.0 / (3.0 * numParticles * kb)) * E_total;
         double lambda = sqrt(temperature / currentTemp);
         for (int i = 0; i < numParticles; i++) {
             U[i] *= lambda;
             V[i] *= lambda;
             W[i] *= lambda;
         }
     }
     
     for (int i = 0; i < numParticles; i++) { //apply boundary conditions
         X[i] += dt * U[i];
         Y[i] += dt * V[i];
         Z[i] += dt * W[i];
         if (X[i] > Lx) {
             X[i] = 2*Lx - X[i];
             U[i] = -fabs(U[i]);
         }
         if (Y[i] > Ly) {
             Y[i] = 2*Ly - Y[i];
             V[i] = -fabs(V[i]);
         }
         if (Z[i] > Lz) {
             Z[i] = 2*Lz - Z[i];
             W[i] = -fabs(W[i]);
         }
         if (X[i] < 0) {
             X[i] = -X[i];
             U[i] = fabs(U[i]);
         }
         if (Y[i] < 0) {
             Y[i] = -Y[i];
             V[i] = fabs(V[i]);
         }
         if (Z[i] < 0) {
             Z[i] = -Z[i];
             W[i] = fabs(W[i]);
         }
     }
 }
 
 
 
 /**
  * @brief Writes simulation data to output files.
  *
  * This function writes particle positions, velocities, kinetic energy, and timestamps to the files energy.txt and positions.txt.
  * energy.txt containes timestamp and kinetic energy of each particle.alignas. eg. Time E1 E2 E3 ... 
  * positions.txt containes the timestamp and x and y position of each particle. eg Time X1 Y1 X2 Y2 X3 Y3...
  * 
  *
  * @param t Current time step index       
  * @param numParticles Number of particles
  * @param X Pointer to the pointer for x coordinates
  * @param Y Pointer to the pointer for y coordinate
  * @param Z Pointer to the pointer for z coordinates
  * @param U Pointer to the pointer for x velocities
  * @param V Pointer to the pointer for y velocities
  * @param W Pointer to the pointer for z velocities
  * @param E Pointer to the pointer that will hold the kinetic energy
  */
 void writeToFiles(int t, int numParticles, const vector<double>& timestamps,
                   const double* X, const double* Y, const double* Z,
                   const double* U, const double* V, const double* W,
                   const double* E)
 {
 
     {
         ofstream energyfile("energy.txt", ios::app);  // write time stamp and KE to kinetic energy file
         energyfile << "runtime";
         for (int i = 0; i < numParticles; i++) {
             energyfile << " E" << i;
         }
         energyfile << "\n";
         energyfile << timestamps[t];
         for (int i = 0; i < numParticles; i++) {
             energyfile << " " << E[i];
         }
         energyfile << "\n";
     }
 
     {
         ofstream posfile("positions.txt", ios::app);   // write time stamp, x and y position to position file
         posfile << "runtime";
         for (int i = 0; i < numParticles; i++) {
             posfile << " x" << i << " y" << i;
         }
         posfile << "\n";
         posfile << std::defaultfloat << timestamps[t];
         for (int i = 0; i < numParticles; i++) {
             posfile << " " << std::fixed << std::setprecision(6) << X[i]
                     << " " << std::fixed << std::setprecision(6) << Y[i];
         }
         posfile << "\n";
     }
 }
 
 /**
  * @brief Main simulation program.
  *
  * This function reads command line arguments, initializes the variables, runs the simulation for each timestep,
  * and writes to output files.
  *
  * @param argc Number of arguments provided
  * @param argv Arguments provided stored as strings
  * @return Exit value
  */
 int main(int argc, char *argv[]) { // read cmd args w main params.
     auto start = chrono::high_resolution_clock::now();              // start runtime clock
     int i = 0;
     double Lx = 20;         // initialise default params
     double Ly = 20;
     double Lz = 20;
     double dt = 0.001;
     bool testCase = false;
     bool timeProvided = false;
     bool nProvided = false;
     bool icRandomChosen = false;
     bool tempProvided = false;
     
     ifstream file1("output.txt"); // close files in case make clean isn't run. Function write to file appends so it's worth doing just in case.
     if (file1) {
         file1.close();
         remove("output.txt");
     }
     ifstream file2("energy.txt");
     if (file2) {
         file2.close();
         remove("energy.txt");
     }
     ifstream file3("positions.txt");
     if (file3) {
         file3.close();
         remove("positions.txt");
     }
     double *X, *Y, *Z, *U, *V, *W, *E, *speed, *Fx, *Fy, *Fz;
     double xij, yij, zij, rij;
 
     map<string, map<string, vector<double>>> testCaseDict = getTestCases();
     
     double runtime, percent_type1, temperature;
     double kb = 0.8314459920816467;
     int numParticles;
     vector<double> x, y, z, u, v, w;
     double* type;  // This will be allocated in variableInitialisation
 
     while (i < argc) {                              // save args given by user into relevant variables
         if (string(argv[i]) == "--Lx") {
             Lx = stod(argv[i + 1]);
         } else if (string(argv[i]) == "--Ly") {
             Ly = stod(argv[i + 1]);
         } else if (string(argv[i]) == "--Lz") {
             Lz = stod(argv[i + 1]);
         } else if (string(argv[i]) == "--T") {
             runtime = stod(argv[i + 1]);
             timeProvided = true;
         } else if (string(argv[i]) == "--N") {
             numParticles = stoi(argv[i + 1]);
             nProvided = true;
         } else if (string(argv[i]) == "--temp") {
             temperature = stod(argv[i + 1]);
             tempProvided = true;
         } else if (string(argv[i]) == "--percent-type1") {
             percent_type1 = stod(argv[i + 1]);
         } else if (string(argv[i]) == "--ic-random") {
             icRandomChosen = true;
         } else if (testCaseDict.find(string(argv[i])) != testCaseDict.end()) {
             string key(argv[i]);
             runtime = testCaseDict[key]["runtime"][0];
             numParticles = testCaseDict[key]["numParticles"][0];
             x = testCaseDict[key]["x"];
             y = testCaseDict[key]["y"];
             z = testCaseDict[key]["z"];
             u = testCaseDict[key]["u"];
             v = testCaseDict[key]["v"];
             w = testCaseDict[key]["w"];
             testCase = true;
         } else if (string(argv[i]) == "--help") {
             cout << "Allowed options:\n"
                  << "--help                Print available options.\n"
                  << "--Lx arg (=20)        x length (Angstroms)\n"
                  << "--Ly arg (=20)        y length (Angstroms)\n"
                  << "--Lz arg (=20)        z length (Angstroms)\n"
                  << "--dt arg (=0.001)     Time-step\n"
                  << "--T arg               Final time\n"
                  << "--ic-one              Initial condition: one stationary particle\n"
                  << "--ic-one-vel          Initial condition: one moving particle\n"
                  << "--ic-two              Initial condition: two bouncing particles\n"
                  << "--ic-two-pass1        Initial condition: two passing particles close\n"
                  << "--ic-two-pass2        Initial condition: two passing particles close\n"
                  << "--ic-two-pass3        Initial condition: two passing particles close, heavy\n"
                  << "--percent-type1 arg (=10)  Percentage of type 1 particles with random IC\n"
                  << "--ic-random           Number of particles to spawn with random IC\n"
                  << "--temp arg            Temperature (degrees Kelvin)\n";
             exit(1);
         }
         i++;
     }
     
     if ((testCase == true) || (icRandomChosen && nProvided && timeProvided)) {  // check if args are valid
         cout << "Command Line input well-formatted, carrying on..." << endl;
     } else {
         cout << "Command line input formatted incorrectly, exiting program." << endl;
         exit(1);
     }
     
     int totalSteps = (runtime / dt) + 1;
     vector<double> timestamps(totalSteps);
     for (int i = 0; i < totalSteps; i++) {
         timestamps[i] = i * dt;  // vector going from 0 to time T in increments dt
     }
 
     // allocate vars in managed memory
     variableInitialisation(totalSteps, numParticles, &X, &Y, &Z, &U, &V, &W, &E, &speed, &Fx, &Fy, &Fz, &type);
 
     if (icRandomChosen) {
         icRandom(numParticles, Lx, Ly, Lz, percent_type1, X, Y, Z, U, V, W, type);
     } else if (testCase == true) {
         for (int i = 0; i < numParticles; i++) {
             X[i] = x[i];
             Y[i] = y[i];
             Z[i] = z[i];
             U[i] = u[i];
             V[i] = v[i];
             W[i] = w[i];
         }
     }
     
     
     int epsilon[2][2] = { {3,15}, {15,60} }; //initialise epsilon and sigma as stated in brief
     int sigma[2][2] = { {1,2}, {2,3} };
 
     constexpr int n = 2048; //compile time constant. n value found to be most optimal through trial and error
     int threads = min(256, n);
     int blocks = max(n/256, 1);
 
     for (int t = 0; t < totalSteps; t++) {
         for (int i = 0; i < numParticles; i++) {
             Fx[i] = 0.0;
             Fy[i] = 0.0;
             Fz[i] = 0.0;           
         }
 
         updateVars<<<blocks, threads>>>(numParticles, dt, Lx, Ly, Lz, type, temperature, tempProvided, kb,
             epsilon, sigma, X, Y, Z, U, V, W, E, speed, Fx, Fy, Fz);  //run CUDA kernal updateVars 
             
         if (t % 100 == 0) {
             writeToFiles(t, numParticles, timestamps,X,Y,Z,U,V, W,E);
         }
         hipDeviceSynchronize(); //wait for GPU ops to complete before next time step iteration
     }
 
     hipFree(X); // release managed memory
     hipFree(Y);
     hipFree(Z);
     hipFree(U);
     hipFree(V);
     hipFree(W);
     hipFree(E);
     hipFree(speed);
     hipFree(Fx);
     hipFree(Fy);
     hipFree(Fz);
     hipFree(type);
     
     auto end = chrono::high_resolution_clock::now();
     chrono::duration<double> duration = end - start;
     cout << "Runtime: " << duration.count() << " seconds" << endl;
     
     return 0;
 }
 