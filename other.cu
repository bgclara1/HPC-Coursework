#include "hip/hip_runtime.h"

 #include "md.cuh"

 // CUDA kernel to compute forces
 _global_
 void compute_forces_kernel(Particle *d_particles, int numParticles, double *d_epsilon, double *d_sigma) {
     int i = blockIdx.x * blockDim.x + threadIdx.x;
     if (i >= numParticles) return;
 
     Particle &pi = d_particles[i];
     pi.fx = 0.0;
     pi.fy = 0.0;
     pi.fz = 0.0;
 
     for (int j = i + 1; j < numParticles; ++j) {
         Particle &pj = d_particles[j];
 
         double dx = pj.x - pi.x;
         double dy = pj.y - pi.y;
         double dz = pj.z - pi.z;
 
         double r2 = dx * dx + dy * dy + dz * dz;
         if (r2 < 0.0001) continue;
 
         int type1 = pi.type;
         int type2 = pj.type;
         double eps = d_epsilon[type1 * 2 + type2];
         double sig = d_sigma[type1 * 2 + type2];
 
         double sig2 = sig * sig;
         double r2_inv = 1.0 / r2;
         double sig_r2_inv = sig2 * r2_inv;
         double r6 = sig_r2_inv * sig_r2_inv * sig_r2_inv;
         double r12 = r6 * r6;
 
         double f = 24.0 * eps * (2.0 * r12 - r6) * r2_inv;
 
         double fx = f * dx;
         double fy = f * dy;
         double fz = f * dz;
 
         pi.fx -= fx;
         pi.fy -= fy;
         pi.fz -= fz;
 
         pj.fx+= fx;
         pj.fy+= fy;
         pj.fz+= fz;
     }
 }
 
 // Host function to manage CUDA force computation
 void compute_forces_gpu() {
     Particle *d_particles;
     double *d_epsilon, *d_sigma;
 
     int numParticles = particles.size();
     size_t particleSize = numParticles * sizeof(Particle);
     size_t matrixSize = 4 * sizeof(double);
 
     // Allocate GPU memory
     hipMalloc((void **)&d_particles, particleSize);
     hipMalloc((void **)&d_epsilon, matrixSize);
     hipMalloc((void **)&d_sigma, matrixSize);
 
     // Copy data to GPU
     hipMemcpy(d_particles, particles.data(), particleSize, hipMemcpyHostToDevice);
     hipMemcpy(d_epsilon, epsilon, matrixSize, hipMemcpyHostToDevice);
     hipMemcpy(d_sigma, sigma, matrixSize, hipMemcpyHostToDevice);
 
     // Define CUDA grid dimensions
     int threadsPerBlock = 256;
     int blocksPerGrid = (numParticles + threadsPerBlock - 1) / threadsPerBlock;
 
     // Launch kernel
     compute_forces_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_particles, numParticles, d_epsilon, d_sigma);
     hipDeviceSynchronize();
 
     // Copy results back to CPU
     hipMemcpy(particles.data(), d_particles, particleSize, hipMemcpyDeviceToHost);
 
     // Free GPU memory
     hipFree(d_particles);
     hipFree(d_epsilon);
     hipFree(d_sigma);
 }
 
 using namespace std;
 void simulate_cuda() {
     const int steps = static_cast<int>(final_time / dt);
     const int output_interval = static_cast<int>(0.1 / dt);
     double global_min_dist = std::numeric_limits<double>::max();
 
     for (int step = 0; step <= steps; ++step) {
         double current_time = step * dt;
         if (step % output_interval == 0) {
             write_output(current_time);
         }
 
         if (current_time != final_time) {
             compute_forces_gpu();
             update_velocities();
             update_positions();
             if (test_case!=0){
                 compute_min_distance(global_min_dist);
 
             }
         }
     }
     if (test_case!=0) {
         ofstream min_dist_file("minimum_distance.txt");
         min_dist_file << "Smallest Distance between any 2 particles across All Time: " << global_min_dist << "\n";
         min_dist_file.close();
         data_file.close();
     }
 }