#include <iostream>
#include <map>
#include <string>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <fstream>
#include <iomanip>
#include <ctime>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

__constant__ double s6_table[4] = {0.0, 1.0, 64.0, 729.0};

void variableInitialisation(int totalSteps, int numParticles,
    double** X, double** Y, double** Z,
    double** U, double** V, double** W,
    double** E, double** speed,
    double** Fx, double** Fy, double** Fz,
    double** type)
{
    size_t size = totalSteps * numParticles * sizeof(double);
    hipMallocManaged((void**)X, size);
    hipMallocManaged((void**)Y, size);
    hipMallocManaged((void**)Z, size);
    hipMallocManaged((void**)U, size);
    hipMallocManaged((void**)V, size);
    hipMallocManaged((void**)W, size);
    hipMallocManaged((void**)E, size);
    hipMallocManaged((void**)speed, size);
    hipMallocManaged((void**)Fx, size);
    hipMallocManaged((void**)Fy, size);
    hipMallocManaged((void**)Fz, size);

    hipMemset(*X, 0, size);
    hipMemset(*Y, 0, size);
    hipMemset(*Z, 0, size);
    hipMemset(*U, 0, size);
    hipMemset(*V, 0, size);
    hipMemset(*W, 0, size);
    hipMemset(*E, 0, size);
    hipMemset(*speed, 0, size);
    hipMemset(*Fx, 0, size);
    hipMemset(*Fy, 0, size);
    hipMemset(*Fz, 0, size);

    size_t typeSize = numParticles * sizeof(double);
    hipMallocManaged((void**)type, typeSize);
    hipMemset(*type, 0, typeSize);
}

void icRandom(int numParticles, double Lx, double Ly, double Lz, double percent_type1,
    double* X, double* Y, double* Z,
    double* U, double* V, double* W,
    double* type)
{
    srand(time(0));

    // Initialize positions and velocities.
    for (int i = 0; i < numParticles; i++) {
        double cx, cy, cz;
        while (true) {
            cx = ((double)rand() / RAND_MAX) * Lx;
            cy = ((double)rand() / RAND_MAX) * Ly;
            cz = ((double)rand() / RAND_MAX) * Lz;
            bool valid = true;
            for (int j = 0; j < i; j++) {
                double dx = cx - X[j];
                double dy = cy - Y[j];
                double dz = cz - Z[j];
                if (dx * dx + dy * dy + dz * dz < 0.25) {  // 0.5^2 = 0.25
                    valid = false;
                    break;
                }
            }
            if (valid)
                break;
        }
        X[i] = cx;
        Y[i] = cy;
        Z[i] = cz;
        U[i] = ((double)rand() / RAND_MAX) - 0.5;
        V[i] = ((double)rand() / RAND_MAX) - 0.5;
        W[i] = ((double)rand() / RAND_MAX) - 0.5;
    }

    int numType1 = (int)ceil(numParticles * (percent_type1 / 100.0));
    for (int i = 0; i < numParticles; i++) {
        type[i] = 0;
    }
    for (int i = 0; i < numType1; i++) {
        type[i] = 1;
    }
    for (int i = 0; i < numParticles; i++) {
        int j = rand() % numParticles;
        double temp = type[i];
        type[i] = type[j];
        type[j] = temp;
    }
}

map<string, map<string, vector<double>>> getTestCases() {
    map<string, map<string, vector<double>>> testCaseDict;
    testCaseDict["--ic-one"] = {
        {"runtime", {0.2}},
        {"numParticles", {1}},
        {"x", {10.0}},
        {"y", {10.0}},
        {"z", {10.0}},
        {"u", {0.0}},
        {"v", {0.0}},
        {"w", {0.0}},
        {"type", {0}}
    };
    testCaseDict["--ic-one-vel"] = {
        {"runtime", {20.0}},
        {"numParticles", {1}},
        {"x", {10.0}},
        {"y", {10.0}},
        {"z", {10.0}},
        {"u", {5.0}},
        {"v", {2.0}},
        {"w", {1.0}},
        {"type", {0}}
    };
    testCaseDict["--ic-two"] = {
        {"runtime", {50}},
        {"numParticles", {2}},
        {"x", {8.5, 11.5}},
        {"y", {10.0, 10.0}},
        {"z", {10.0, 10.0}},
        {"u", {0.0, 0.0}},
        {"v", {0.0, 0.0}},
        {"w", {0.0, 0.0}},
        {"type", {0, 0}}
    };
    testCaseDict["--ic-two-pass1"] = {
        {"runtime", {50.0}},
        {"numParticles", {2}},
        {"x", {8.5, 11.5}},
        {"y", {11.5, 8.5}},
        {"z", {10.0, 10.0}},
        {"u", {0.5, -0.5}},
        {"v", {0.0, 0.0}},
        {"w", {0.0, 0.0}},
        {"type", {0, 0}}
    };
    testCaseDict["--ic-two-pass2"] = {
        {"runtime", {50.0}},
        {"numParticles", {2}},
        {"x", {8.5, 11.5}},
        {"y", {11.3, 8.7}},
        {"z", {10.0, 10.0}},
        {"u", {0.5, -0.5}},
        {"v", {0.0, 0.0}},
        {"w", {0.0, 0.0}},
        {"type", {0, 0}}
    };
    testCaseDict["--ic-two-pass3"] = {
        {"runtime", {50.0}},
        {"numParticles", {2}},
        {"x", {8.5, 11.5}},
        {"y", {11.3, 8.7}},
        {"z", {10.0, 10.0}},
        {"u", {0.5, -0.5}},
        {"v", {0.0, 0.0}},
        {"w", {0.0, 0.0}},
        {"type", {1, 1}}
    };
    return testCaseDict;
}

__global__
void updateVars(int numParticles, double dt, double Lx, double Ly, double Lz,
    double* type, double temperature, bool tempProvided, double kb,
    const int epsilon[2][2], const int sigma[2][2],
    double* X, double* Y, double* Z,
    double* U, double* V, double* W,
    double* E, double* speed, double* Fx, double* Fy, double* Fz)
{

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= numParticles) return;
    for (int i = 0; i < numParticles; i++) {
        for (int j = i + 1; j < numParticles; j++) {
            double xij = X[i] - X[j];
            double yij = Y[i] - Y[j];
            double zij = Z[i] - Z[j];
            double rij = xij*xij + yij*yij + zij*zij; // r squared
            int t1 = static_cast<int>(type[i]);
            int t2 = static_cast<int>(type[j]);
            int e = epsilon[t1][t2];
            int s = sigma[t1][t2];

            double inv_r4 = 1.0 / (rij * rij * rij * rij);
            double sigma6_val = s6_table[s] * inv_r4;
            double sigma12 = sigma6_val * sigma6_val * rij;
            double coeff = -24.0 * e * (2.0 * sigma12 - sigma6_val);

            Fx[i] -= xij * coeff;
            Fy[i] -= yij * coeff;
            Fz[i] -= zij * coeff;
            Fx[j] += xij * coeff;
            Fy[j] += yij * coeff;
            Fz[j] += zij * coeff;
        }
    }
    
    for (int i = 0; i < numParticles; i++) {
        int m = (type[i] == 0) ? 1 : 10;
        U[i] += dt * Fx[i] / m;
        V[i] += dt * Fy[i] / m;
        W[i] += dt * Fz[i] / m;
    }
    
    double E_total = 0.0;
    for (int i = 0; i < numParticles; i++) {
        int m = (type[i] == 0) ? 1 : 10;
        double speed2 = U[i]*U[i] + V[i]*V[i] + W[i]*W[i];
        E[i] = 0.5 * m * speed2;
        E_total += E[i];
    }
    
    if (tempProvided) {
        double currentTemp = (2.0 / (3.0 * numParticles * kb)) * E_total;
        double lambda = sqrt(temperature / currentTemp);
        for (int i = 0; i < numParticles; i++) {
            U[i] *= lambda;
            V[i] *= lambda;
            W[i] *= lambda;
        }
    }
    
    for (int i = 0; i < numParticles; i++) {
        X[i] += dt * U[i];
        Y[i] += dt * V[i];
        Z[i] += dt * W[i];
        if (X[i] > Lx) {
            X[i] = 2*Lx - X[i];
            U[i] = -fabs(U[i]);
        }
        if (Y[i] > Ly) {
            Y[i] = 2*Ly - Y[i];
            V[i] = -fabs(V[i]);
        }
        if (Z[i] > Lz) {
            Z[i] = 2*Lz - Z[i];
            W[i] = -fabs(W[i]);
        }
        if (X[i] < 0) {
            X[i] = -X[i];
            U[i] = fabs(U[i]);
        }
        if (Y[i] < 0) {
            Y[i] = -Y[i];
            V[i] = fabs(V[i]);
        }
        if (Z[i] < 0) {
            Z[i] = -Z[i];
            W[i] = fabs(W[i]);
        }
    }
}

void writeToFiles(int t, int numParticles, const vector<double>& timestamps,
                  const double* X, const double* Y, const double* Z,
                  const double* U, const double* V, const double* W,
                  const double* E)
{

    {
        ofstream outfile("output.txt", ios::app);
        outfile << "Time step " << t << "\n";
    }

    {
        ofstream energyfile("energy.txt", ios::app);
        energyfile << "runtime";
        for (int i = 0; i < numParticles; i++) {
            energyfile << " E" << i;
        }
        energyfile << "\n";
        energyfile << timestamps[t];
        for (int i = 0; i < numParticles; i++) {
            energyfile << " " << E[i];
        }
        energyfile << "\n";
    }

    {
        ofstream posfile("positions.txt", ios::app);
        posfile << "runtime";
        for (int i = 0; i < numParticles; i++) {
            posfile << " x" << i << " y" << i;
        }
        posfile << "\n";
        posfile << std::defaultfloat << timestamps[t];
        for (int i = 0; i < numParticles; i++) {
            posfile << " " << std::fixed << std::setprecision(6) << X[i]
                    << " " << std::fixed << std::setprecision(6) << Y[i];
        }
        posfile << "\n";
    }
}

int main(int argc, char *argv[]) {
    auto start = chrono::high_resolution_clock::now();
    int i = 0;
    double Lx = 20, Ly = 20, Lz = 20, dt = 0.001;
    bool testCase = false, timeProvided = false, nProvided = false, icRandomChosen = false, tempProvided = false;

    ifstream file1("output.txt"); if (file1) { file1.close(); remove("output.txt"); }
    ifstream file2("energy.txt"); if (file2) { file2.close(); remove("energy.txt"); }
    ifstream file3("positions.txt"); if (file3) { file3.close(); remove("positions.txt"); }

    double *X, *Y, *Z, *U, *V, *W, *E, *speed, *Fx, *Fy, *Fz;
    double xij, yij, zij, rij;

    map<string, map<string, vector<double>>> testCaseDict = getTestCases();
    
    double runtime, percent_type1, temperature;
    double kb = 0.8314459920816467;
    int numParticles;
    vector<double> x, y, z, u, v, w;
    double* type;  // This will be allocated in variableInitialisation

    while (i < argc) {
        if (string(argv[i]) == "--Lx") { Lx = stod(argv[i+1]); }
        else if (string(argv[i]) == "--Ly") { Ly = stod(argv[i+1]); }
        else if (string(argv[i]) == "--Lz") { Lz = stod(argv[i+1]); }
        else if (string(argv[i]) == "--T") { runtime = stod(argv[i+1]); timeProvided = true; }
        else if (string(argv[i]) == "--N") { numParticles = stoi(argv[i+1]); nProvided = true; }
        else if (string(argv[i]) == "--temp") { temperature = stod(argv[i+1]); tempProvided = true; }
        else if (string(argv[i]) == "--percent-type1") { percent_type1 = stod(argv[i+1]); }
        else if (string(argv[i]) == "--dt") { dt = stod(argv[++i]); }
        else if (string(argv[i]) == "--ic-random") { icRandomChosen = true; }
        else if (testCaseDict.find(string(argv[i])) != testCaseDict.end()) {
            string key(argv[i]);
            runtime = testCaseDict[key]["runtime"][0];
            numParticles = testCaseDict[key]["numParticles"][0];
            x = testCaseDict[key]["x"];
            y = testCaseDict[key]["y"];
            z = testCaseDict[key]["z"];
            u = testCaseDict[key]["u"];
            v = testCaseDict[key]["v"];
            w = testCaseDict[key]["w"];
            // Instead of assigning a pointer from a temporary vector, we'll copy later.
            testCase = true;
        }
        else if (string(argv[i]) == "--help") {
            cout << "Allowed options:\n"
                 << "--help                Print available options.\n"
                 << "--Lx arg (=20)        x length (Angstroms)\n"
                 << "--Ly arg (=20)        y length (Angstroms)\n"
                 << "--Lz arg (=20)        z length (Angstroms)\n"
                 << "--dt arg (=0.001)     Time-step\n"
                 << "--T arg               Final time\n"
                 << "--ic-one              Initial condition: one stationary particle\n"
                 << "--ic-one-vel          Initial condition: one moving particle\n"
                 << "--ic-two              Initial condition: two bouncing particles\n"
                 << "--ic-two-pass1        Initial condition: two passing particles close\n"
                 << "--ic-two-pass2        Initial condition: two passing particles close\n"
                 << "--ic-two-pass3        Initial condition: two passing particles close, heavy\n"
                 << "--percent-type1 arg (=10)  Percentage of type 1 particles with random IC\n"
                 << "--ic-random           Number of particles to spawn with random IC\n"
                 << "--temp arg            Temperature (degrees Kelvin)\n";
            exit(1);
        }
        i++;
    }
    
    if ((testCase == true) || (icRandomChosen && nProvided && timeProvided)) {
        cout << "Command Line input well-formatted, carrying on..." << endl;
    } else {
        cout << "Command line input formatted incorrectly, exiting program." << endl;
        exit(1);
    }
    
    int totalSteps = (runtime / dt) + 1;
    vector<double> timestamps(totalSteps);
    for (int i = 0; i < totalSteps; i++) {
        timestamps[i] = i * dt;
    }

    // Allocate all arrays (including type) in managed memory.
    variableInitialisation(totalSteps, numParticles, &X, &Y, &Z, &U, &V, &W, &E, &speed, &Fx, &Fy, &Fz, &type);

    if (icRandomChosen) {
        icRandom(numParticles, Lx, Ly, Lz, percent_type1, X, Y, Z, U, V, W, type);
    } else if (testCase == true) {
        for (int i = 0; i < numParticles; i++) {
            X[i] = x[i];
            Y[i] = y[i];
            Z[i] = z[i];
            U[i] = u[i];
            V[i] = v[i];
            W[i] = w[i];
            //type[i] = type[i];
        }
    }
    
    
    int epsilon[2][2] = { {3,15}, {15,60} };
    int sigma[2][2] = { {1,2}, {2,3} };

    constexpr int n = 2048;
    int threads = min(256, n);
    int blocks = max(n/256, 1);

    for (int t = 0; t < totalSteps; t++) {
        for (int i = 0; i < numParticles; i++) {
            Fx[i] = 0.0;
            Fy[i] = 0.0;
            Fz[i] = 0.0;           
        }

        updateVars<<<blocks, threads>>>(numParticles, dt, Lx, Ly, Lz, type, temperature, tempProvided, kb,
            epsilon, sigma, X, Y, Z, U, V, W, E, speed, Fx, Fy, Fz);
            
        if (t % 10 == 0) {
            writeToFiles(t, numParticles, timestamps,X,Y,Z,U,V, W,E);
        }
        hipDeviceSynchronize();
    }

    hipFree(X);
    hipFree(Y);
    hipFree(Z);
    hipFree(U);
    hipFree(V);
    hipFree(W);
    hipFree(E);
    hipFree(speed);
    hipFree(Fx);
    hipFree(Fy);
    hipFree(Fz);
    hipFree(type);
    
    auto end = chrono::high_resolution_clock::now();
    chrono::duration<double> duration = end - start;
    cout << "Runtime: " << duration.count() << " seconds" << endl;
    
    return 0;
}
